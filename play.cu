
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

using namespace std;

struct S {
    
    double a[100];
    double max;

} s;


__global__ void findMax(S* d_out, S* d_in){
 
    double m = d_in->a[0];

    for (int i = 0; i < 100; i++) {

        if (*(d_in->a + i) > m) {
            m = *(d_in->a + i);
        }

    }

    d_out->max = m;

    
}


int main(){


    for(int i = 0; i < 100; i++){
        s.a[i] = i + 0.6;
    }


    //host memory
    S* h_in = &s;
    S* h_out = new S;

    size_t memory_size = sizeof(s);

    //Declare and allocate device memory
    S* d_in;
    S* d_out;
    hipMalloc((void**)&d_in, memory_size); 
    hipMalloc((void**)&d_out, memory_size);

    hipMemcpy(d_in, h_in, memory_size, hipMemcpyHostToDevice);//data transfer from host to device

    findMax<<<1, 1>>>(d_out, d_in);//Kernel function, calculate each particle's density. 4 blocks in total; 625 threads per block.
    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, memory_size, hipMemcpyDeviceToHost); //data transfer from device to host

    hipFree(d_in); hipFree(d_out);//free allocated device memory

    s = *h_out; //get the contents of coordinates in host's end

    cout << s.max << endl;

    return 0;

}